#include "hip/hip_runtime.h"
#include "MathUtils.h"
#include "glm/geometric.hpp"
#include <hip/hip_runtime.h>

namespace Math {
    __host__ __device__ glm::vec3 FromSphericalCoords(float theta, float phi) {
        return {
            glm::sin(theta) * glm::cos(phi),
            glm::sin(theta) * glm::sin(phi),
            glm::cos(theta)
        };
    }

    __host__ __device__ glm::vec3 ToWorldSpace(const glm::vec3 &local, const glm::vec3 &normal) {
        glm::vec3 up;
        if (abs(normal.x) < c_INV_SQRT3) {
            up = glm::vec3(1, 0, 0);
        } else if (abs(normal.y) < c_INV_SQRT3) {
            up = glm::vec3(0, 1, 0);
        } else {
            up = glm::vec3(0, 0, 1);
        }

        glm::vec3 tangent = glm::normalize(glm::cross(normal, up));
        glm::vec3 bitangent = glm::normalize(glm::cross(normal, tangent));
        return tangent * local.x + bitangent * local.y + normal * local.z;
    }

    __host__ __device__ glm::vec3 SampleUniformHemisphere(const glm::vec3 &normal, const glm::vec2 &uniform) {
        float z = uniform.x;
        float r = sqrtf(std::max(0.0f, 1.0f - z*z));

        float phi = 2 * c_PI * uniform.y;
        glm::vec3 local = glm::vec3(r*cosf(phi), r*sinf(phi), z);
        return ToWorldSpace(local, normal);
    }

    __host__ __device__ glm::vec3 SampleCosineHemisphere(const glm::vec3 &normal, const glm::vec2 &uniform) {        
        float z = sqrtf(uniform.x);
        
        float phi = 2 * c_PI * uniform.y;
        float r = sqrtf(std::max(0.0f, 1.0f - z*z));
        glm::vec3 local = glm::vec3(r*cosf(phi), r*sinf(phi), z);
        return ToWorldSpace(local, normal);
    }

    __host__ __device__ glm::vec3 Reflect(const glm::vec3 &vec, const glm::vec3 &normal) {
        return 2 * glm::dot(vec, normal) * normal - vec;
    }

    __host__ __device__ bool Refract(const glm::vec3 &vec, const glm::vec3 &normal, float eta, glm::vec3 &result) {
        float cos_in = glm::dot(normal, vec);
        glm::vec3 n = normal;
        
        if (cos_in < 0) {
            eta = 1.0f / eta;
            cos_in = -cos_in;
            n = -n;
        }
        float sin2_in = fmaxf(0.f, 1.f - cos_in * cos_in);
        float sin2_t = sin2_in * eta * eta;
        if (sin2_t >= 1.f) {
            // Total internal reflection
            return false;
        }
        float cos_t = fmaxf(0.0f, sqrtf(1.0f - sin2_t));

        result = -vec * eta + (cos_in * eta - cos_t) * n;
        return true;
    }

}
