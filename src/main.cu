#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <volk.h>

__global__ void HelloCuda(int i, int b) {
    int a = i + b;
    printf("Hello, CUDA!\n");
}

int main() {
    HelloCuda<<<1,1>>>(1, 1);

    hipDeviceSynchronize();
    return 0;
}